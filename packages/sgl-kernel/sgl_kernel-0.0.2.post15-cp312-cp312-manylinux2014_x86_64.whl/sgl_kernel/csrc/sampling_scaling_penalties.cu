#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <THC/THCAtomics.cuh>

#include "utils.hpp"
#include "vectorization.cuh"

template <typename scalar_t>
__global__ void sampling_scaling_penalties_kernel(const scalar_t* logits, const scalar_t* scaling_penalties,
                                                  scalar_t* output, const int32_t numel) {
  const int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int32_t stride = blockDim.x * gridDim.x;

  auto const* vectorized_logits = reinterpret_cast<vec4_t<scalar_t> const*>(logits);
  auto const* vectorized_penalties = reinterpret_cast<vec4_t<scalar_t> const*>(scaling_penalties);
  auto* vectorized_output = reinterpret_cast<vec4_t<scalar_t>*>(output);

  const int32_t num_vec_elems = numel >> 2;

#pragma unroll 4
  for (int32_t i = tid; i < num_vec_elems; i += stride) {
    vec4_t<scalar_t> logits_vec = vectorized_logits[i];
    vec4_t<scalar_t> penalties_vec = vectorized_penalties[i];
    vec4_t<scalar_t> out_vec;

    out_vec.x = logits_vec.x > 0 ? logits_vec.x / penalties_vec.x : logits_vec.x * penalties_vec.x;
    out_vec.y = logits_vec.y > 0 ? logits_vec.y / penalties_vec.y : logits_vec.y * penalties_vec.y;
    out_vec.z = logits_vec.z > 0 ? logits_vec.z / penalties_vec.z : logits_vec.z * penalties_vec.z;
    out_vec.w = logits_vec.w > 0 ? logits_vec.w / penalties_vec.w : logits_vec.w * penalties_vec.w;

    vectorized_output[i] = out_vec;
  }

  const int32_t start_idx = num_vec_elems * 4;
  for (int32_t i = start_idx + tid; i < numel; i += stride) {
    scalar_t logit = logits[i];
    scalar_t penalty = scaling_penalties[i];
    output[i] = logit > 0 ? logit / penalty : logit * penalty;
  }
}

torch::Tensor sampling_scaling_penalties(const torch::Tensor& logits, const torch::Tensor& scaling_penalties) {
  auto output = torch::empty_like(logits);
  const auto numel = logits.numel();
  const int threads = 512;

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, logits.scalar_type(), "sampling_scaling_penalties_kernel", ([&] {
        const int blocks = (numel + threads * 4 - 1) / (threads * 4);
        sampling_scaling_penalties_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            logits.data_ptr<scalar_t>(), scaling_penalties.data_ptr<scalar_t>(), output.data_ptr<scalar_t>(), numel);
      }));

  return output;
}
