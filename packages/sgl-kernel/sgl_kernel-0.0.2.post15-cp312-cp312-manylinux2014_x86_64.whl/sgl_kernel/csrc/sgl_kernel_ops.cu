#include <vector>

#include "utils.hpp"

// trt_reduce
using fptr_t = int64_t;
fptr_t init_custom_ar(int64_t rank_id, int64_t world_size, torch::Tensor& rank_data, const std::vector<fptr_t>& buffers,
                      const std::vector<fptr_t>& tmp_result_buffers, const std::vector<fptr_t>& barrier_in,
                      const std::vector<fptr_t>& barrier_out);
void dispose(fptr_t _fa);
void all_reduce(fptr_t _fa, torch::Tensor& inp, torch::Tensor& out);
std::tuple<std::vector<int64_t>, std::vector<int64_t>> get_graph_buffer_ipc_meta(fptr_t _fa);
void register_graph_buffers(fptr_t _fa, const std::vector<std::vector<int64_t>>& handles,
                            const std::vector<std::vector<int64_t>>& offsets);

// moe_align_block_size
void moe_align_block_size(torch::Tensor topk_ids, int64_t num_experts, int64_t block_size,
                          torch::Tensor sorted_token_ids, torch::Tensor experts_ids, torch::Tensor num_tokens_post_pad,
                          torch::Tensor token_cnts_buffer, torch::Tensor cumsum_buffer);

// sampling_scaling_penalties
torch::Tensor sampling_scaling_penalties(const torch::Tensor& logits, const torch::Tensor& scaling_penalties);

// int8_scaled_mm
torch::Tensor int8_scaled_mm(const torch::Tensor& mat_a, const torch::Tensor& mat_b, const torch::Tensor& scales_a,
                             const torch::Tensor& scales_b, const torch::Dtype& out_dtype,
                             const c10::optional<torch::Tensor>& bias);

// rotary embedding
void rotary_embedding(torch::Tensor& positions, torch::Tensor& query, torch::Tensor& key, int64_t head_size,
                      torch::Tensor& cos_sin_cache, bool is_neox);

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  // trt_reduce
  m.def("init_custom_ar", &init_custom_ar, "init custom allreduce meta (CUDA)");
  m.def("dispose", &dispose, "dispose custom allreduce meta");
  m.def("all_reduce", &all_reduce, "custom all reduce (CUDA)");
  m.def("get_graph_buffer_ipc_meta", &get_graph_buffer_ipc_meta, "custom all reduce get graph ipc meta");
  m.def("register_graph_buffers", &register_graph_buffers, "custom all reduce register graph buffers");
  // moe_align_block_size
  m.def("moe_align_block_size", &moe_align_block_size, "MOE Align Block Size (CUDA)");
  // sampling_scaling_penalties
  m.def("sampling_scaling_penalties", &sampling_scaling_penalties, "Sampling scaling penalties (CUDA)");
  // int8_scaled_mm
  m.def("int8_scaled_mm", &int8_scaled_mm, "INT8 scaled matmul (CUDA)");
  // rotary embedding
  m.def("rotary_embedding", &rotary_embedding, "Rotary Embedding (CUDA)");
}
